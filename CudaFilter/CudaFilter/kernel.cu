#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"
#include <iostream>

__global__
void filter(unsigned char* input_image, unsigned char* output_image, int width, int height) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < 2 || col < 2 || row >= height - 3 || col >= width - 3)
		return;

	// GAUSS BLUR / 16
	float kernel[3][3] = {
		{ 1, 2, 1 },
		{ 2, 4, 2 },
		{ 1, 2, 1 }
	};

	int sum = 0;
	int kernelSum = 0;
	for (int j = -1; j <= 1; j++) {
		for (int i = -1; i <= 1; i++)
		{
			int pixel = input_image[(row + j) * width + (col + i)];
			int val = pixel * kernel[i + 1][j + 1];
			sum += val;
			kernelSum += kernel[i][j];
		}
	}

	int value = sum / (kernelSum);
	output_image[row * width + col] = (value > 0) ? value : 0;
}


void convolution(unsigned char* input_image, unsigned char* output_image, int width, int height) {

	unsigned char* dev_input;	// declare arrays for pixels from image
	unsigned char* dev_output;	// same

	hipMalloc((void**)&dev_input, width*height * sizeof(unsigned char));	// Alocate memory on cuda with size of image, each pixel is char with color value(num)
	hipMalloc((void**)&dev_output, width*height * sizeof(unsigned char));	// Alocate memory for image output

	hipMemcpy(dev_input, input_image, width*height * sizeof(unsigned char), hipMemcpyHostToDevice);	// Copying input array, pixels of image on cuda memory

	dim3 blockDims(32, 32);	// define size of blocks which we sent to cuda thread
	dim3 gridDims(width / 32, height / 32);	// define grid, grid and blocks define size of each data on each cuda thread

	filter <<< gridDims, blockDims >>> (dev_input, dev_output, width, height);	// Call filter method for each thread, defined by block and grid size
	hipDeviceSynchronize();	// waiting for all threads to finish job

	hipMemcpy(output_image, dev_output, width*height * sizeof(unsigned char), hipMemcpyDeviceToHost);	// copy the output image(array of pixels) from cuda memory to cpu memory

	hipFree(dev_input);	// free arrays
	hipFree(dev_output);
}